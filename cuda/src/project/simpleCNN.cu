#include "hip/hip_runtime.h"
// ���򻯰���

// ��Ŀ7���򻯰�������
// ����: ��дһ��CUDA����ʵ��һ���򻯰��2D���������

// Ҫ��:

// ��ʼ��һ��MxN����������һ��С��KxK����˾���
// ʹ��CUDA�ں˺���������������
// ��ӡ����������


#include <cmath>
#include <hip/hip_runtime.h>
#include <>
#include "simpleCNN.cuh"




__global__ void convolution2D(float* input, float* kernel, float* output, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int kernelRadius = k / 2;

    if (row < m && col < n) {
        float value = 0.0f;
        for (int i = -kernelRadius; i <= kernelRadius; ++i) {
            for (int j = -kernelRadius; j <= kernelRadius; ++j) {
                int curRow = row + i;
                int curCol = col + j;
                if (curRow >= 0 && curRow < m && curCol >= 0 && curCol < n) {
                    value += input[curRow * n + curCol] * kernel[(i + kernelRadius) * k + (j + kernelRadius)];
                }
            }
        }
        output[row * n + col] = value;
    }
}

void printMatrix(const std::vector<float>& matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}


std::vector<float>  run(std::vector<float> input, std::vector<float> kernel, size_t rows, size_t cols, size_t kernelSize) {
    
    size_t M = rows;  // ������������
    size_t N = cols;  // ������������
    size_t K = kernelSize;  // ����˾���Ĵ�С

    std::vector<float> output(M * N, 0);

    float* d_input, * d_kernel, * d_output;

    hipMalloc(&d_input, M * N * sizeof(float));
    hipMalloc(&d_kernel, K * K * sizeof(float));
    hipMalloc(&d_output, M * N * sizeof(float));

    hipMemcpy(d_input, input.data(), M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel.data(), K * K * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    convolution2D << <gridSize, blockSize >> > (d_input, d_kernel, d_output, M, N, K);
    hipDeviceSynchronize();

    hipMemcpy(output.data(), d_output, M * N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Input Matrix:" << std::endl;
    printMatrix(input, M, N);

    std::cout << "Kernel Matrix:" << std::endl;
    printMatrix(kernel, K, K);

    std::cout << "Output Matrix:" << std::endl;
    printMatrix(output, M, N);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);

    return output;
}
