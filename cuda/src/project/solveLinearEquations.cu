  // 8��������Է�����
  // ��Ŀ8��������Է����飨Ax = b��
  // ����: ��дһ��CUDA����������Է�����Ax = b��

  // Ҫ��:

  // ��ʼ��һ����СΪNxN�ľ���A��һ����СΪN������b��
  // ʹ��CUDA�ں˺�����ⷽ���飬�õ�����x��
  // ��ӡ����������x��



#include "solveLinearEquations.cuh"



#define CHECK_CUDA(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl; \
            exit(-1); \
        } \
    } while (0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS Error: " << status << std::endl; \
            exit(-1); \
        } \
    } while (0)

void solveLinearSystem(const std::vector<float>& A, const std::vector<float>& b, std::vector<float>& x, int n) {
    float* d_A, * d_b;
    int* d_pivot, * d_info;

    // Allocate device memory
    CHECK_CUDA(hipMalloc((void**)&d_A, n * n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_b, n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_pivot, n * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_info, sizeof(int)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, A.data(), n * n * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, b.data(), n * sizeof(float), hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    int lda = n;

    // Perform LU factorization
    float* Aarray[] = { d_A };
    CHECK_CUBLAS(hipblasSgetrfBatched(handle, n, Aarray, lda, d_pivot, d_info, 1));

    // Solve Ax = b using the LU factorization
    float* Barray[] = { d_b };
    CHECK_CUBLAS(hipblasSgetrsBatched(handle, HIPBLAS_OP_N, n, 1, (const float**)Aarray, lda, d_pivot, Barray, lda, d_info, 1));

    // Copy the result back to host
    CHECK_CUDA(hipMemcpy(x.data(), d_b, n * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_b));
    CHECK_CUDA(hipFree(d_pivot));
    CHECK_CUDA(hipFree(d_info));

    CHECK_CUBLAS(hipblasDestroy(handle));
}

std::vector<float> runSolveLinearEquations(std::vector<float> A_, std::vector<float> b_, int n) {
  
 /*   const int n = 3;*/

    std::vector<float> A = {
        1.0f, 2.0f, 3.0f,
        4.0f, 5.0f, 6.0f,
        7.0f, 8.0f, 10.0f
    };

    std::vector<float> b = { 6.0f, 15.0f, 25.0f };

    std::vector<float> x(n, 0.0f);

    solveLinearSystem(A, b, x, n);

    std::cout << "Solution: ";
    for (int i = 0; i < n; ++i) {
        std::cout << x[i] << " ";
    }
    std::cout << std::endl;


    return x;
}
