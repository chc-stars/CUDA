#include "hip/hip_runtime.h"
// 10.����ǰ׺��(ɨ��)

// ��Ŀ10������ǰ׺�ͣ�ɨ�裩
// ����: ��дһ��CUDA����ʵ������Ĳ���ǰ׺�ͣ�ɨ�裩���㡣

// Ҫ��:

// ��ʼ��һ����СΪN������A��
// ʹ��CUDA�ں˺������������ǰ׺������B��ʹ��B[i] = A[0] + A[1] + ... + A[i]��
// ��ӡ����������B��

#include "src/common/error.cuh"
#include "hip/hip_runtime.h"
#include ""


__global__ void scan_up_sweep(int* d_out, int N, int stride) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int k = idx * stride * 2;

    if (k + stride < N) {
        d_out[k + stride * 2 - 1] += d_out[k + stride - 1];
    }
}


__global__ void scan_down_sweep(int* d_out, int N, int stride) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int k = idx * stride * 2;
    if (k + stride < N) {
        int temp = d_out[k + stride - 1];
        d_out[k + stride - 1] = d_out[k + stride * 2 - 1];
        d_out[k + stride * 2 - 1] += temp;

    }

}

void prefix_sum(int* h_out, const int* h_in, int N) {

    int* d_out;
    size_t size = N * sizeof(int);
    CHECK(hipMalloc((void**)&d_out, size));
    CHECK(hipMemcpy(d_out, h_in, size, hipMemcpyHostToDevice));

    int threads = 512;
    int blocks = (N + threads * 1) / threads;

    // ����ɨ��
    for (int stride = 1; stride < N; stride *= 2) {
        scan_up_sweep << <blocks, threads >> > (d_out, N, stride);

    }

    hipMemset(&d_out[N - 1], 0, sizeof(int));

    // ����ɨ��
    for (int stride = N / 2; stride >= 1; stride /= 2) {
        scan_down_sweep << <blocks, threads >> > (d_out, N, stride);
    }

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    hipFree(d_out);
}

int runParallelPrefixSum(int h_in[], int h_out[], int N) {

  

    // ����ǰ׺��
    prefix_sum(h_out, h_in, N);

    // ��ӡ������
    std::cout << "Input: ";
    for (int i = 0; i < N; ++i) {
        std::cout << h_in[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Output: ";
    for (int i = 0; i < N; ++i) {
        std::cout << h_out[i] << " ";
    }
    std::cout << std::endl;

    return 0;

}