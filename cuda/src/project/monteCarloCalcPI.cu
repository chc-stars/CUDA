#include "hip/hip_runtime.h"
// 6�����ؿ��޼���pI

// ��Ŀ6�����ؿ��޷��������ֵ
// ����: ��дһ��CUDA����ʹ�����ؿ��޷�������е�ֵ��

// Ҫ��:

// ��ʼ��һ����СΪN�����飬���ڴ洢����㡣
// ʹ��CUDA�ں˺��������ڵ�λԲ�ڵĵ��������
// ���㲢��ӡ����еĽ���ֵ��

#include <hip/hip_runtime.h>
#include "src/common/error.cuh"
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <>

#define N 1000
#define BLOCK_SIZE 256

__global__ void monteCarloPiKernel(int* count, unsigned int seed) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState state;

    hiprand_init(seed, idx, 0, &state);

    float x = hiprand_uniform(&state);
    float y = hiprand_uniform(&state);

    if (x * x + y * y <= 1.0f) {
        atomicAdd(count, 1);
    }

}


void calculatePi() {

    int* d_count;
    int h_count = 0;

    CHECK(hipMalloc(&d_count, sizeof(int)));
    CHECK(hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice));

    int gridSize = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);
    monteCarloPiKernel << <gridSize, BLOCK_SIZE >> > (d_count, time(NULL));
    CHECK(hipDeviceSynchronize());

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float elaspedTime;
    hipEventElapsedTime(&elaspedTime, start, end);

    std::cout << "Time: " << elaspedTime << std::endl;

    CHECK(hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_count));

    float pi = 4.0f * h_count / N;
    std::cout << "Estimated Pi = " << pi << std::endl;
}

int runCalculatePi() {
    calculatePi();
    return 0;
}