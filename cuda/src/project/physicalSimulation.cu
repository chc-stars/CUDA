#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include ""

#include "src/project/physicalSimulation.cuh"



//#define N  1024 // grid Size
#define BLOCK_SIZE 256  

__global__ void updateGrid(float *grid, float *newGrid, int n, float dt, float dx) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > 0 && idx < n - 1) {
		float left = grid[idx - 1];
		float right = grid[idx + 1];
		float center = grid[idx];

		newGrid[idx] = center + dt * (left - 2 * center + right) / (dx * dx);

	}
}


void runPhysicalSimulation(float *h_grid,  int N) {

	//float *h_grid = new float[N];
	float *h_newGrid = new float[N];
	float *d_grid, *d_newGrid;

	// init grid
	//for (int i = 0; i < N; ++i) {
	//	h_grid[i] = 0.0f;
	//}
	//h_grid[N / 2] = 1.0f;   // ��ʼ����

	// 
	hipMalloc((void**)&d_grid, N * sizeof(float));
	hipMalloc((void**)&d_newGrid, N * sizeof(float));

	// �����ݴ��������䵽�豸
	hipMemcpy(d_grid, h_grid, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_newGrid, h_newGrid, N * sizeof(float), hipMemcpyHostToDevice);

	// ��������Ϳ��С
	dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 threads(BLOCK_SIZE);

	// ʱ�䲽���Ϳռ䲽��
	float dt = 0.01f;
	float dx = 1.0f;

	// ��ѭ��
	for (int step = 0; step < 1000; ++step) {
		updateGrid << <blocks, threads >> > (d_grid, d_newGrid, N, dt, dx);
		// ����ָ��
		float* temp = d_grid;
		d_grid = d_newGrid;
		d_newGrid = temp;

	}

	// ������豸��������
	hipMemcpy(h_grid, d_grid, N * sizeof(float), hipMemcpyDeviceToHost);

	// �ͷ��豸�ڴ�
	hipFree(d_grid);
	hipFree(d_newGrid);


	for (int i = 0; i < N; ++i) {
		std::cout << "Grid[" << i << "] = " << h_grid[i] << std::endl;
	}

	delete[] h_grid;
	delete[] h_newGrid;

}