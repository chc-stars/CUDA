#include "hip/hip_runtime.h"
// 2������˷�

// ��Ŀ2������˷�
// ����: ��дһ��CUDA����ʵ����������ĳ˷���

// Ҫ��:

// ��ʼ��������СΪMxN��NxP�ľ���A��B��
// ʹ��CUDA�ں˺����������˷�C = A * B��
// ��ӡ����������C��

#include "src/common/error.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <>



__global__ void matrixMulti(float* a, float* b, float* c, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}


int runMatrixMulti(float a[], float b[], float c[], size_t N) {


    // �����豸������ָ��
    float* d_a, * d_b, * d_c;

    // �����豸���ڴ�
    CHECK(hipMalloc((void**)&d_a, N * sizeof(float)));
    CHECK(hipMalloc((void**)&d_b, N * sizeof(float)));
    CHECK(hipMalloc((void**)&d_c, N * sizeof(float)));

    // ���ݿ������豸��
    CHECK(hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice));

    // �����߳̿������Ĵ�С
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // ����CUDA�ں�
    matrixMulti << <gridSize, blockSize >> > (d_a, d_b, d_c, N);

    // ͬ���豸��
    CHECK(hipDeviceSynchronize());

    // ��������豸�˸��Ƶ�������
    CHECK(hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost));

    // �ͷ��豸���ڴ�
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));

    // ��ӡ���
    for (int i = 0; i < N; ++i) {
        printf("c[%d] = %f\n", i, c[i]);
    }

    return 0;

}

