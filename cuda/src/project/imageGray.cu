#include "hip/hip_runtime.h"

// ��Ŀ4��ͼ��ҶȻ�
// ����: ��дһ��CUDA���򣬽�һ��RGBͼ��ת��Ϊ�Ҷ�ͼ��

// Ҫ��:

// ��ʼ��һ��MxNx3��RGBͼ��
// ʹ��CUDA�ں˺�������Ҷ�ͼ�񣬻Ҷ�ֵ���㹫ʽΪGray = 0.299R + 0.587G + 0.114*B��
// ��ӡ�������Ҷ�ͼ��
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include ""

// Time to run on CPU: 5381.61 ms
// Gray Image (CPU):
// Time to run kernel: 7.72288 ms
// Gray Image (GPU):

#define M 2600 // ͼ��߶�
#define N 2600 // ͼ����

__global__ void rgb2gray(unsigned char* rgb, unsigned char* gray, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        unsigned char r = rgb[3 * idx];
        unsigned char g = rgb[3 * idx + 1];
        unsigned char b = rgb[3 * idx + 2];
        gray[idx] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}

void initializeImage(unsigned char* image, int width, int height) {
    for (int i = 0; i < width * height * 3; i++) {
        image[i] = rand() % 256; // �����ʼ��ͼ������
    }
}

void printImage(unsigned char* image, int width, int height, int channels) {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            for (int k = 0; k < channels; k++) {
                printf("%d ", image[(i * width + j) * channels + k]);
            }
            printf(" | ");
        }
        printf("\n");
    }
}

void rgb2grayCPU(unsigned char* rgb, unsigned char* gray, int width, int height) {
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            int idx = y * width + x;
            unsigned char r = rgb[3 * idx];
            unsigned char g = rgb[3 * idx + 1];
            unsigned char b = rgb[3 * idx + 2];
            gray[idx] = 0.299f * r + 0.587f * g + 0.114f * b;
        }
    }
}

int runImgGray() {
    int width = N;
    int height = M;
    size_t rgb_size = width * height * 3 * sizeof(unsigned char);
    size_t gray_size = width * height * sizeof(unsigned char);

    // ���������ڴ�
    unsigned char* h_rgb = (unsigned char*)malloc(rgb_size);
    unsigned char* h_gray = (unsigned char*)malloc(gray_size);
    unsigned char* h_gray_cpu = (unsigned char*)malloc(gray_size);

    // ��ʼ��RGBͼ��
    initializeImage(h_rgb, width, height);

    // ��ӡRGBͼ��
    std::cout << "RGB Image:\n";
    // printImage(h_rgb, width, height, 3);

    // CPU ����Ҷ�ͼ�񲢲���ʱ��
    auto start_cpu = std::chrono::high_resolution_clock::now();
    rgb2grayCPU(h_rgb, h_gray_cpu, width, height);
    auto stop_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration_cpu = stop_cpu - start_cpu;
    std::cout << "Time to run on CPU: " << duration_cpu.count() << " ms" << std::endl;

    // ��ӡ�Ҷ�ͼ��CPU ��������
    std::cout << "Gray Image (CPU):\n";
    // printImage(h_gray_cpu, width, height, 1);

    // �����豸�ڴ�
    unsigned char* d_rgb, * d_gray;
    hipMalloc((void**)&d_rgb, rgb_size);
    hipMalloc((void**)&d_gray, gray_size);

    // �������ݵ��豸
    hipMemcpy(d_rgb, h_rgb, rgb_size, hipMemcpyHostToDevice);

    // �����������С
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // ����CUDA�¼�
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ��¼��ʼ�¼�
    hipEventRecord(start, 0);

    // ����CUDA�ں�
    rgb2gray << <gridSize, blockSize >> > (d_rgb, d_gray, width, height);

    // ��¼�����¼�
    hipEventRecord(stop, 0);

    // ͬ���¼�
    hipEventSynchronize(stop);

    // ����ʱ���
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time to run kernel: " << elapsedTime << " ms" << std::endl;

    // �����Ҷ�ͼ�����ݻ�����
    hipMemcpy(h_gray, d_gray, gray_size, hipMemcpyDeviceToHost);

    // ��ӡ�Ҷ�ͼ��
    std::cout << "Gray Image (GPU):\n";
    // printImage(h_gray, width, height, 1);

    // �ͷ�CUDA�¼�
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // �ͷ��豸�ڴ�
    hipFree(d_rgb);
    hipFree(d_gray);

    // �ͷ������ڴ�
    free(h_rgb);
    free(h_gray);
    free(h_gray_cpu);

    return 0;
}
