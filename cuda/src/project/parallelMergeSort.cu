#include "hip/hip_runtime.h"
// 5�����й鲢����

// ��Ŀ5�����й鲢����
// ����: ��дһ��CUDA����ʵ�ֲ��й鲢�����㷨��

// Ҫ��:

// ��ʼ��һ����СΪN����������A��
// ʹ��CUDA�ں˺������еض�������й鲢����
// ��ӡ������������顣

#include "src/common/error.cuh"

#include <>
#include "src/project/parallelMergeSort.cuh"


__device__ void merge(int* arr, int* temp, int left, int mid, int right) {
    int i = left;
    int j = mid + 1;
    int k = left;

    while (i <= mid && j <= right) {
        if (arr[i] <= arr[j]) {
            temp[k++] = arr[i++];
        }
        else {
            temp[k++] = arr[j++];
        }
    }

    while (i <= mid) {
        temp[k++] = arr[i++];
    }

    while (j <= right) {
        temp[k++] = arr[j++];
    }

    for (i = left; i <= right; i++) {
        arr[i] = temp[i];
    }
}

__global__ void mergeSortKernel(int* arr, int* temp, int width, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int left = idx * width * 2;
    int mid = min(left + width - 1, size - 1);
    int right = min(left + 2 * width - 1, size - 1);

    if (left < size && mid < size) {
        merge(arr, temp, left, mid, right);
    }
}

void mergeSort(int* arr, int size) {
    int* d_arr, * d_temp;
    size_t bytes = size * sizeof(int);

    hipMalloc(&d_arr, bytes);
    hipMalloc(&d_temp, bytes);
    hipMemcpy(d_arr, arr, bytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    for (int width = 1; width < size; width *= 2) {
        mergeSortKernel << <gridSize, blockSize >> > (d_arr, d_temp, width, size);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr, bytes, hipMemcpyDeviceToHost);

    hipFree(d_arr);
    hipFree(d_temp);
}

void initializeArray(int* arr, int size) {
    for (int i = 0; i < size; i++) {
        arr[i] = rand() % 100 * 5;
    }
}

void printArray(int* arr, int size) {
    for (int i = 0; i < size; i++) {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}

std::vector<int> runParallelMergeSort(std::vector<int> arr) {

    int size = arr.size();
    initializeArray(arr.data(), size);

    std::cout << "Unsorted array:" << std::endl;
    printArray(arr.data(), size);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);
    mergeSort(arr.data(), size);

    hipEventRecord(end, 0);

    // ͬ���¼�
    hipEventSynchronize(end);


    // ����ʱ���
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, end);
    std::cout << "Time to run kernel: " << elapsedTime << " ms" << std::endl;

    std::cout << "Sorted array:" << std::endl;

    printArray(arr.data(), size);

    return arr;
}



