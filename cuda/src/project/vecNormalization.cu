#include "hip/hip_runtime.h"
// // 3. ������һ��

// ��Ŀ3��������һ��
// ����: ��дһ��CUDA����ʵ�������Ĺ�һ��������������ÿ��Ԫ�س��������ĳ��ȣ���

// Ҫ��:

// ��ʼ��һ����СΪN������A��
// ʹ��CUDA�ں˺��������һ������B��ʹ��B[i] = A[i] / ||A||������||A||������A��ŷ����÷�������
// ��ӡ����������B��

#include <hip/hip_runtime.h>
#include "src/common/error.cuh"
#include <iostream>
#include <cmath>
#include <>



__global__ void vecNorm(const float* a, float* b, int n) {
    __shared__ float sum[256]; // �����ڴ�����
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    // ����ÿ��Ԫ�ص�ƽ��
    sum[tid] = (idx < n) ? a[idx] * a[idx] : 0.0f;

    __syncthreads();

    // ��Լ���
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sum[tid] += sum[tid + stride];
        }
        __syncthreads();
    }

    // ����Լ���д��ȫ���ڴ�
    if (tid == 0) {
        atomicAdd(&b[0], sum[0]);
    }
}

__global__ void normalize(float* a, float* b, float norm, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        b[idx] = a[idx] / norm;
    }
}

int runVecNormalization(float a[], float b[],  int N) {
 
    float h_sum = 0.0f;
  

    float* d_a, * d_b;

    // �����ڴ�
    CHECK(hipMalloc((void**)&d_a, N * sizeof(float)));
    CHECK(hipMalloc((void**)&d_b, N * sizeof(float)));

    // copy����
    CHECK(hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, &h_sum, sizeof(float), hipMemcpyHostToDevice));

    // ����block
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // �����ں˼���������С
    vecNorm << <gridSize, blockSize >> > (d_a, d_b, N);
    CHECK(hipDeviceSynchronize());

    // ��ƽ���ʹ��豸���Ƶ�����
    CHECK(hipMemcpy(&h_sum, d_b, sizeof(float), hipMemcpyDeviceToHost));

    // ����������L2����
    float norm = sqrt(h_sum);

    // ��һ������
    normalize << <gridSize, blockSize >> > (d_a, d_a, norm, N);
    CHECK(hipDeviceSynchronize());

    // ��������豸���Ƶ�����
    CHECK(hipMemcpy(b, d_a, N * sizeof(float), hipMemcpyDeviceToHost));

    // �ͷ��豸�ڴ�
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));

    // ��ӡ���
    for (int i = 0; i < N; i++) {
        printf("b[%d] = %f\n", i, b[i]);
    }

    return 0;
}
